
#include "engine/kernels.cuh"
#include "engine/multi-column-host-kernels.cuh"
#include "generated-bindings/kernel-bindings.cuh"
#include <stdexcept>

namespace bindings {

template <>
uint64_t*
decompress_column<uint64_t, flsgpu::device::FFORColumn<uint64_t>>(const flsgpu::device::FFORColumn<uint64_t> column,
                                                                  const unsigned        unpack_n_vectors,
                                                                  const unsigned        unpack_n_values,
                                                                  const enums::Unpacker unpacker,
                                                                  const enums::Patcher  patcher,
                                                                  const uint32_t        n_samples) {
	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 1, 1, flsgpu::device::FFORFunctor<uint64_t, 1>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>>(column, n_samples);
	}

	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 1, 1, flsgpu::device::FFORFunctor<uint64_t, 1>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>>(column, n_samples);
	}
	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 4, 1, flsgpu::device::FFORFunctor<uint64_t, 4>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>>(column, n_samples);
	}

	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 4, 1, flsgpu::device::FFORFunctor<uint64_t, 4>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>>(column, n_samples);
	}
	throw std::invalid_argument("Could not find correct binding in decompress_column FFOR<uint64_t>");
}

} // namespace bindings
