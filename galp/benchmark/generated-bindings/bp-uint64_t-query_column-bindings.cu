
#include "engine/kernels.cuh"
#include "engine/multi-column-host-kernels.cuh"
#include "generated-bindings/kernel-bindings.cuh"
#include <stdexcept>

namespace bindings {

template <>
bool query_column<uint64_t, flsgpu::device::BPColumn<uint64_t>>(const flsgpu::device::BPColumn<uint64_t> column,
                                                                const unsigned        unpack_n_vectors,
                                                                const unsigned        unpack_n_values,
                                                                const enums::Unpacker unpacker,
                                                                const enums::Patcher  patcher,
                                                                const uint64_t        magic_value,
                                                                const uint32_t        n_samples) {
	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::query_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 1, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, magic_value, n_samples);
	}

	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::query_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 1, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, magic_value, n_samples);
	}
	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::query_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 4, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, magic_value, n_samples);
	}

	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::query_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 4, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, magic_value, n_samples);
	}
	throw std::invalid_argument("Could not find correct binding in query_column BP<uint64_t>");
}

} // namespace bindings
