
#include "engine/kernels.cuh"
#include "engine/multi-column-host-kernels.cuh"
#include "generated-bindings/kernel-bindings.cuh"
#include <stdexcept>

namespace bindings {

template <>
bool compute_column<uint64_t, flsgpu::device::FFORColumn<uint64_t>>(const flsgpu::device::FFORColumn<uint64_t> column,
                                                                    const unsigned        unpack_n_vectors,
                                                                    const unsigned        unpack_n_values,
                                                                    const enums::Unpacker unpacker,
                                                                    const enums::Patcher  patcher,
                                                                    const unsigned        n_repetitions,
                                                                    const uint32_t        n_samples) {
	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::compute_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 1, 1, flsgpu::device::FFORFunctor<uint64_t, 1>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>,
		    10>(column, n_samples);
	}

	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::compute_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 1, 1, flsgpu::device::FFORFunctor<uint64_t, 1>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>,
		    10>(column, n_samples);
	}
	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::compute_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 4, 1, flsgpu::device::FFORFunctor<uint64_t, 4>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>,
		    10>(column, n_samples);
	}

	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::compute_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::FFORDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 4, 1, flsgpu::device::FFORFunctor<uint64_t, 4>>,
		        flsgpu::device::FFORColumn<uint64_t>>,
		    flsgpu::device::FFORColumn<uint64_t>,
		    10>(column, n_samples);
	}
	throw std::invalid_argument("Could not find correct binding in compute_column FFOR<uint64_t>");
}

} // namespace bindings
