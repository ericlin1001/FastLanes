// ────────────────────────────────────────────────────────
// |                      FastLanes                       |
// ────────────────────────────────────────────────────────
// galp/benchmark/generated-bindings/bp-uint64_t-decompress_column-bindings.cu
// ────────────────────────────────────────────────────────

#include "engine/kernels.cuh"
#include "engine/multi-column-host-kernels.cuh"
#include "generated-bindings/kernel-bindings.cuh"
#include <stdexcept>

namespace bindings {

template <>
uint64_t*
decompress_column<uint64_t, flsgpu::device::BPColumn<uint64_t>>(const flsgpu::device::BPColumn<uint64_t> column,
                                                                const unsigned        unpack_n_vectors,
                                                                const unsigned        unpack_n_values,
                                                                const enums::Unpacker unpacker,
                                                                const enums::Patcher  patcher,
                                                                const uint32_t        n_samples) {
	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 1, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, n_samples);
	}

	if (unpack_n_vectors == 1 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    1,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        1,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 1, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, n_samples);
	}
	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::Dummy &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerDummy<uint64_t, 4, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, n_samples);
	}

	if (unpack_n_vectors == 4 && unpack_n_values == 1 && unpacker == enums::Unpacker::StatefulBranchless &&
	    patcher == enums::Patcher::None) {
		return kernels::host::decompress_column<
		    uint64_t,
		    4,
		    1,
		    flsgpu::device::BPDecompressor<
		        uint64_t,
		        4,
		        flsgpu::device::BitUnpackerStatefulBranchless<uint64_t, 4, 1, flsgpu::device::BPFunctor<uint64_t>>,
		        flsgpu::device::BPColumn<uint64_t>>,
		    flsgpu::device::BPColumn<uint64_t>>(column, n_samples);
	}
	throw std::invalid_argument("Could not find correct binding in decompress_column BP<uint64_t>");
}

} // namespace bindings
