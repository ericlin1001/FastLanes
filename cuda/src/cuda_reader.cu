// ────────────────────────────────────────────────────────
// |                      FastLanes                       |
// ────────────────────────────────────────────────────────
// cuda/src/cuda_reader.cu
// ────────────────────────────────────────────────────────

#include <hip/hip_runtime.h>
#include <cstdio>

// A simple CUDA Hello World example
// This kernel prints a greeting from each thread on the GPU
__global__ void hello_from_gpu() {
	int tid = threadIdx.x;
	printf("Hello World from GPU! Thread %d says hi.\n", tid);
}

int main() {
	// Launch the kernel with 4 threads in one block
	hello_from_gpu<<<1, 4>>>();

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// And return
	return 0;
}
